#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>
#include <cstdlib>
#include <chrono>
#include <cstdlib>
#include <algorithm>
#include <thrust/transform_reduce.h>
#include <cmath>
#include <random>
#include <thrust/device_vector.h>

using namespace std::chrono;

void reportTime(const char* msg, steady_clock::duration span) {
    auto ms = duration_cast<milliseconds>(span);
    std::cout << msg << " - levou - " <<
    ms.count() << " milisegundos" << std::endl;
}

// CRIE UMA FUNCTOR PARA CALCULAR A SQUARE

struct calculate_square
{
    __host__ __device__
    double operator()(const double& x) {
        return x * x;
    }
};


// IMPLEMENTE O CALCULO DA MAGNITUDE COM THRUST
float magnitude(thrust::device_vector<double>& vector_norm) {
    float result;

    // ... add Thrust calls
    // AQUI VAO AS CHAMADAS THRUST 

    calculate_square unary_op;
    thrust::plus<float> binary_op;
    float init=0;

    result = std::sqrt(thrust::transform_reduce(vector_norm.begin(), vector_norm.end(), unary_op,init, binary_op));

    return result;
}

int main(int argc, char** argv) {
    if (argc != 2) {
        std::cerr << argv[0] << ": numero invalido de argumentos\n"; 
        std::cerr << "uso: " << argv[0] << "  tamanho do vetor\n"; 
        return 1;
    }
    int n = std::atoi(argv[1]); //numero de elementos
    steady_clock::time_point ts, te;

    // Faça um  vector em thrust 

    thrust::device_vector<double> vector_norm (n);

    // inicilize o  vector
    ts = steady_clock::now();
    std::generate(vector_norm.begin(), vector_norm.end(), rand);
    te = steady_clock::now();
    reportTime("Inicializacao", te - ts);

    // Calcule a magnitude do vetor
    ts = steady_clock::now();
    float len = magnitude(vector_norm);
    te = steady_clock::now();
    reportTime("Tempo para calculo", te - ts);


    std::cout << std::fixed << std::setprecision(4);
    std::cout << "Magnitude : " << len << std::endl;
}