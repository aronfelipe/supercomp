#include "hip/hip_runtime.h"
#include <thrust/transform.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/iterator/counting_iterator.h>
#include <iostream>
#include <thrust/copy.h>

using namespace std;
using namespace thrust;

// struct saxpy
// {
//     // int a;    
//     // saxpy(int a_) : a(a_) {};
//     __host__ __device__
//     char operator()(const char& x, const char& y) {
//         printf("X : %s\n", x);
//         printf("Y : %s\n", y);

//         return x;

//         //    return a * x + y;
//     }
// };


// struct calculate_diagonal_insertion
// {
//     char letter;
//     char *data_subsequence;
//     int *last_line_calculation;

//     calculate_diagonal_insertion(char *_data_subsequence, char _letter, int *_last_line_calculation) : data_subsequence(_data_subsequence), letter(_letter), last_line_calculation(_last_line_calculation) {};

//     // thrust::device_vector<char> data_subsequence_final(data_subsequence.begin(), data_subsequence.end());

//     __host__ __device__
//     int operator()(const int& counting_i) {

//         // thrust::device_vector<double> v3(data_subsequence, data_subsequence + 10);
//         int i = counting_i;

//         // printf("v3[i]: %d\n", v3[i]);

//         int diagonal = 0;

//         if (data_subsequence[i] == letter) {
//             diagonal = last_line_calculation[i-1] + 2;
//         } else {
//             diagonal = last_line_calculation[i-1] - 1;
//         }

//         int insertion = last_line_calculation[i] - 1;
//         printf("last line : %d\n", last_line_calculation[i-1]);
//         // printf("last line : %d\n", data_subsequence[i]);

//         // printf("insertion : %d\n", insertion);

//         if (diagonal > insertion) {
//             if (diagonal > 0) {
//                 return diagonal;
//             } else {
//                 return 0;
//             }
//         } else {
//             if (insertion > 0) {
//                 // printf("PASSOU : %d\n", insertion);

//                 return insertion;
//             } else {
//                 return 0;
//             }
//         }

//     }
// };

struct calculate_diagonal_insertion
{
    char letter;
    char *data_subsequence;
    // int *last_line_calculation;

    calculate_diagonal_insertion(char *_data_subsequence, char _letter) : data_subsequence(_data_subsequence), letter(_letter){};

    // thrust::device_vector<char> data_subsequence_final(data_subsequence.begin(), data_subsequence.end());

    __host__ __device__
    int operator()(const int& counting_i) {

        // thrust::device_vector<double> v3(data_subsequence, data_subsequence + 10);
        int i = counting_i;

        // printf("v3[i]: %d\n", v3[i]);

        printf("data subsequence[i]: %d\n", data_subsequence[i]);

        int diagonal = 0;
        return diagonal;
        // if (data_subsequence[i] == letter) {
        //     diagonal = last_line_calculation[i-1] + 2;
        // } else {
        //     diagonal = last_line_calculation[i-1] - 1;
        // }

        // int insertion = last_line_calculation[i] - 1;
        // printf("last line : %d\n", last_line_calculation[i-1]);
        // printf("last line : %d\n", data_subsequence[i]);

        // printf("insertion : %d\n", insertion);

        // if (diagonal > insertion) {
        //     if (diagonal > 0) {
        //         return diagonal;
        //     } else {
        //         return 0;
        //     }
        // } else {
        //     if (insertion > 0) {
        //         // printf("PASSOU : %d\n", insertion);

        //         return insertion;
        //     } else {
        //         return 0;
        //     }
        // }

    }
};


// struct calculate_deletion
// {


//     // calculate_deletion() : data_subsequence(_data_subsequence), letter(_letter), last_line_calculation(_last_line_calculation) {};

//     __host__ __device__
//     int operator()(const int& i_1, const int& i) {
//         printf("I : %d\n", i);
//         printf("I - 1 : %d\n", i_1);

//         return i;
//         // // printf("J : %d\n", j);

//         // return i_1;

//         // if (i_1 - 1 > i)

//         // printf(counting_i);
//         // printf("Call for value : %d\n", counting_i);
//         // int i = counting_i;
//         // // int j = position_letter;
//         // int diagonal = 0;
//         // if (data_subsequence[i] == letter) {
//         //     diagonal = last_line_calculation[i-1] + 2;
//         // } else {
//         //     diagonal = last_line_calculation[i-1] - 1;
//         // }
//         // printf("Call for value : %d\n", diagonal);
//         // int insertion = last_line_calculation[i] - 1;
        
//         // int max_between_diagonal_insertion = std::max(diagonal, insertion);
//         // int max_between_diagonal_insertion_zero = std::max(max_between_diagonal_insertion, 0);

//         // return diagonal;
//     }
// };


int main() {

    double n = 0;
    double m = 0;

    cin >> n >> m;

    host_vector<char> dnas_first_seq_cpu;
    host_vector<char> dnas_second_seq_cpu;

    dnas_first_seq_cpu.reserve(n);
    dnas_second_seq_cpu.reserve(m);

    char dna_next;

    // Reads the first sequence and completes the dna_first_seq vector with the sequence.
    for(int i = 0; i < n; i++) {
        cin >> dna_next;
        dnas_first_seq_cpu.push_back(dna_next);
    }

    // Reads the second sequence and completes the dna_second_seq vector with the sequence.
    for(int i = 0; i < m; i++) {
        cin >> dna_next;
        dnas_second_seq_cpu.push_back(dna_next);
    }
    
    int k = n;
    int l = m;

    host_vector<char> dnas_first_seq_cpu_flexible = dnas_first_seq_cpu;
    host_vector<char> dnas_second_seq_cpu_flexible = dnas_second_seq_cpu;

    host_vector<host_vector<char> > subsequences_a;
    host_vector<host_vector<char> > subsequences_b;

    while (dnas_first_seq_cpu_flexible.size() > 0) {
        while(k > 0) {
            host_vector<char> next_subsequence;
            for (int j = 0; j < k; j++) {
                next_subsequence.push_back(dnas_first_seq_cpu_flexible[j]);
            }
            k = k - 1;

            subsequences_a.push_back(next_subsequence);
            next_subsequence.clear();
        }
        dnas_first_seq_cpu_flexible.erase(dnas_first_seq_cpu_flexible.begin());
        k = dnas_first_seq_cpu_flexible.size();
    }

    while (dnas_second_seq_cpu_flexible.size() > 0) {
        while(l > 0) {
            host_vector<char> next_subsequence;
            for (int j = 0; j < l; j++) {
                next_subsequence.push_back(dnas_second_seq_cpu_flexible[j]);
            }
            l = l - 1;

            subsequences_b.push_back(next_subsequence);
            next_subsequence.clear();
        }
        dnas_second_seq_cpu_flexible.erase(dnas_second_seq_cpu_flexible.begin());
        l = dnas_second_seq_cpu_flexible.size();
    }
    
    int max_score_global = 0;

    for (int i = 0; i < subsequences_a.size(); i++) {
        char *subsequence_from_sequence_a = subsequences_a[i].data();
        device_vector<char> subsequence_a_gpu = subsequences_a[i];
        // int length_of_subsequence_a = strlen(subsequence_from_sequence_a);
        device_vector<int> calculation_temp(subsequences_a[i].size() + 1);
        device_vector<int> calculation_score(subsequences_a[i].size() + 1);

        // int * calculation_temp_data = calculation_temp.data();

        // calculation[0].resize(subsequences_a[i].size()+1);
        // calculation[1].resize(subsequences_a[i].size()+1);
        thrust::fill(calculation_temp.begin(), calculation_temp.end(), 0);

        int max_score_subsequences = 0;

        for (int j = 0; j < subsequences_b.size(); j++) {
            char *subsequence_from_sequence_b = subsequences_a[j].data();
            // int length_of_subsequence_b = strlen(subsequence_from_sequence_b);
            device_vector<char> subsequence_b_gpu = subsequences_b[j];

            // device_vector<char> data_subsequence_a;
            // thrust::copy(subsequences_a[i].begin(), subsequences_a[i].end(), data_subsequence_a);
            for (int t = 0; t < subsequences_b[j].size(); t++) {
                // cout << subsequence_from_sequence_b;
                // cout << endl;

                char letter_from_subsequence_b = subsequences_b[j][t];
                // cout << letter_from_subsequence_b;
                // cout << endl;
                // for(int i = 0; i < subsequence_a_gpu.size(); i++) {
                //     cout << subsequence_a_gpu[i];
                // }
                // cout << endl;

                // for(int i = 0; i < calculation[0].size(); i++) {
                //     cout << calculation[0][i];
                // }
                thrust::host_vector<int> a(10);
                thrust::host_vector<int> b(10);
                thrust::host_vector<int> c(10);
                thrust::generate(a.begin(), a.end(), rand);
                thrust::generate(b.begin(), b.end(), rand);

                //transferimos para a GPU
                // thrust::device_vector<int> d_a = a;
                // thrust::device_vector<int> d_b = b;
                thrust::device_vector<int> counting(subsequence_a_gpu.size() + 1);
                
                for (int q = 1; q < subsequence_a_gpu.size() + 1; q++) {
                    counting[q] = q;
                }

                // thrust::transform(subsequence_a_gpu.begin(), subsequence_a_gpu.end(),
                //                 subsequence_b_gpu.begin(), subsequence_b_gpu.end(),
                //                 saxpy());

                // thrust::transform(d_a.begin(), d_a.end(),
                //     d_b.begin(), d_b.end(),
                //     saxpy());

                // thrust::transform(counting.begin(), counting.end(), calculation_temp.begin() + 1, calculate_diagonal_insertion(&thrust::raw_pointer_cast(subsequence_a_gpu.data()), letter_from_subsequence_b, &thrust::raw_pointer_cast(calculation_temp.data())));
                // thrust::transform(counting.begin(), counting.end(), calculation_temp.begin() + 1, calculate_diagonal_insertion(subsequence_from_sequence_a, letter_from_subsequence_b, calculation_temp.data()));
                thrust::transform(counting.begin(), counting.end(), calculation_temp.begin() + 1, calculate_diagonal_insertion(subsequence_from_sequence_a, letter_from_subsequence_b));

                // thrust::transform(calculation_score.begin() + 1, calculation_score.end(), calculation_temp.begin() + 1, calculation_temp.end(), calculate_deletion());
                // thrust::transform(calculation_score.begin() + 1, calculation_score.end(), calculation_temp.begin() + 1, calculate_deletion());
                // thrust::inclusive_scan(calculation_score.begin() + 1, calculation_score.end(), calculation_temp.begin() + 1, thrust::maximum<int>());

                // cout << calculation[1].size();
                // cout << endl;
                // for(int i = 0; i < calculation_score.size(); i++) {
                //     cout << calculation_score[i];
                //     cout << endl;
                // }

                int max = *max_element(calculation_temp.begin(), calculation_temp.end());

                if (max > max_score_global) {
                    for (auto& element: subsequences_a[i]) {
                        cout << element;
                    }
                    cout << endl;
                    for (auto& element: subsequences_b[j]) {
                        cout << element;
                    }
                    cout << endl;

                    max_score_global = max;
                }


                // cout << endl;
            }

            // cout << max_score_global;
            // if (max_score_subsequences > max_score_global) {
            //     max_score_global = max_score_subsequences;
            // }


        }
    }

    cout << max_score_global;
    // cout << subsequences_a.size();
    // cout << endl;
    // cout << subsequences_b.size();



    // for (int i = 0; i < subsequences_a.size(); i++) {
    //     device_vector<int> s_temp(subsequences_a[i].size());
    //     device_vector<int> s_temp_final(subsequences_a[i].size());
    //     for (int j = 0; j < subsequences_b.size(); j++) {
    //         for (int t = 0; )
            
    //         if (i == 0) {
    //             for (int t = 0; t < subsequences_a[i].size(); t++) {
    //                 s_temp[t] = 0;
    //             }
    //         }
    //         if (j == 0) {
    //             s_temp[0] = 0;
    //         }
    //         // for (auto& el: s_temp) {
    //         //     cout << el;
    //         // }
    //         for(int t = 0; t < s_temp.size(); t++) {
    //             cout << s_temp[t];
    //         }
    //         cout << endl;
    //     }
    // }

    // if (subsequences_a.size() >= subsequences_b.size()) {
    //     for(int i = 0; i < subsequences_a.size(); i++) {
    //         for (int j = 0; j < subsequences_b.size(); j++) {
    //             for (int t = 0; t < subsequences_a[i].size(); t++) {
    //                 cout << subsequences_b[i][t];
    //                 for (int y = 0; y < subsequences_b[j].size(); y++) {
    //                     cout << subsequences_b[j][y];
    //                     cout << endl;

    //                     break;

    //                 }
    //             }
    //             // cout << endl;


    //             // cout << endl;
    //         }
    //     }
    // }

    // for (int i = 0; i < subsequences_a.size(); i++) {

    // }


    // for (auto& el: subsequences_a) {
    //     for (auto& element : el) {
    //         cout << element;
    //     }
    //     cout << endl;
    // }

    // for (auto& el: subsequences_b) {
    //     for (auto& element : el) {
    //         cout << element;
    //     }
    //     cout << endl;
    // }

    // device_vector<device_vector<char> > subsequences_a_gpu = subsequences_a;
    // device_vector<device_vector<char> > subsequences_b_gpu = subsequences_b;


    // for (int i = 0; i){
    //     host_vector<char> next_subsequence;

    // }



    // for (int i = 0; i < (1 << n); ++i) {
    //     string item;
    //     for (j = 0; j < n; ++j) {
    //         if (i & (1 << j)) {
    //             item += sequence[j];
    //         }
    //     }
    //     result.push_back(item);
    // }


    // int sum = 0;

    // host_vector<host_vector<char> > subsequences_a;

    // host_vector<char> dnas_first_seq_cpu_flexible = dnas_first_seq_cpu;

    // int k = dnas_first_seq_cpu.size();

    // while(dnas_first_seq_cpu_flexible.size() > 0) {
    //     host_vector<char> next_subsequence;
    //     for (int i = 0; i < k; i++) {
    //         next_subsequence.push_back(dnas_first_seq_cpu_flexible[i]);
    //     }
    //     subsequences_a.push_back(next_subsequence);
    //     k = dnas_first_seq_cpu_flexible.size() - 1;
    //     dnas_first_seq_cpu_flexible.erase(dnas_first_seq_cpu_flexible.begin());
    // }

    // for (auto& el: subsequences_a) {
    //     for (auto& element : el) {
    //         cout << element;

    //     }
    //     cout << endl;
    // }

    // for(int i = 0; i < dnas_first_seq_cpu.size(); i++) {
    //     host_vector<char> next_subsequence;
    //     next_subsequence.push_back(dnas_first_seq_cpu[])

    // }

    // while(dnas_first_seq_cpu_flexible.size() > 0) {
    //     for (int i = 0; i < dnas_first_seq_cpu_flexible.size(); i++) {
    //         for (int j = 0; j <= i; j++) {
    //             cout << dnas_first_seq_cpu[j];
    //         }
    //         // cout << endl;
    //     }
    //     dnas_first_seq_cpu_flexible.erase(dnas_first_seq_cpu_flexible.begin());
    // }

    // for (int i = 0; i < dnas_first_seq_cpu.size(); i++) {
    //     cout << dnas_first_seq_cpu[i];
    // }

    // for(int i = 0; i < dnas_first_seq_cpu_flexible.size(); i++) {
    //     for()
    // }




    // for(int i = 0; i < dnas_first_seq_cpu.size(); i++) {
    //     for(int j = i; j < dnas_first_seq_cpu.size(); j++) {
    //         next_subsequence.push_back(dnas_first_seq_cpu[j]);
    //     }
    //     subsequences_a.push_back(next_subsequence);
    //     next_subsequence.clear();
    // }

    // for(auto& el: subsequences_a) {
    //     for(auto& element: el) {
    //         cout << element;
    //     }
    //     cout << endl;
    // }

    // while(dnas_first_seq_flexible.size() - k + 1 > 0) {
    //     for (int i = 0; i < k; i++) {
    //         subsequences_a[counter].push_back(dnas_first_seq_flexible[i].dna_value);
    //     }
    //     counter++;
    //     dnas_first_seq_flexible.erase(dnas_first_seq_flexible.begin());
    // }

    // device_vector<device_vector<char> > subsequences_a_gpu = subsequences_a;

    // for(auto& el: subsequences_a_gpu) {
    //     for(auto& element: el) {
    //         cout << element;
    //     }
    //     cout << endl;
    // }

    // cout << sum;


    // int subsequence_size = 1;

    // for (int i = dnas_first_seq_cpu.size(); i > 1; i--) {
    //     for (int j = 0; j < subsequence_size; j++) {
    //         subsequences_a.push_back()
    //         // sum += 1;
    //     }
    // }


    // for (int i = 0; i < dnas_first_seq_cpu.size(); i++) {
    //     while(subsequence_size < dnas_first_seq_cpu.size()) {
    //         next_subsequence.push_back(i)
    //     }
    // }


    // while (counter < dnas_first_seq_cpu.size()) {
    //     for (int i = 0; i < counter; i++) {
    //         next_subsequence.push_back(dnas_first_seq_cpu[i]);
    //     }
    // }
    
    // device_vector<char> subsequences_a(sum);

    // cout << sum;

    // for (int i = dnas_first_seq_cpu.size(); i > 1; i--) {
    //     for (int j = 0; j < dnas_first_seq_cpu.size(); j++) {
    //         sum += 1;
    //     }
    // } 




    // for (int i = 0; i < n; i++) {
    //     cout << dnas_first_seq_cpu[i] << endl;
    // }

    // for (auto& el : dnas_first_seq_cpu) {
    //     cout << el;
    // }

    // dnas_first_seq_gpu = dnas_first_seq_cpu;
    // dnas_second_seq_gpu = dnas_second_seq_cpu;

    // device_vector<char> dnas_first_seq_gpu_flexible = dnas_first_seq_gpu;
    // device_vector<char> dnas_second_seq_gpu_flexible = dnas_second_seq_gpu;

    // int counter = 0;

    // int counter_outside = dnas_first_seq_gpu_flexible.size();

    // while(counter_outside > 0) {
    //     for (int j = 0; j < dnas_first_seq_gpu_flexible.size(); j++) {
    //         for (int i = 0; i < dnas_first_seq_gpu_flexible.size(); i++) {
    //             // subsequences_a[counter].push_back(dnas_first_seq_gpu[i]);
    //         }
    //         counter++;
    //         dnas_first_seq_gpu_flexible.erase(dnas_first_seq_gpu_flexible.begin());
    //     }
    //     dnas_first_seq_gpu_flexible = dnas_first_seq_gpu;
    //     counter_outside--;
    // }

    // double sum = 0;

    // cout << n << endl;

    // for(double i = 1; i < n + 1; i++) {

    //     cout << i << endl;
        
    //     cout << n / i << endl;

    //     sum += n / i;
    // }

    // cout << sum;


    
    


    // Como tratar s1 e s2 de tamanho diferente?

    // Como gerar simultaneamente N pares?


    return 0;
}