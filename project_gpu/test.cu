#include "hip/hip_runtime.h"
#include <thrust/transform.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/iterator/counting_iterator.h>
#include <iostream>
#include <thrust/copy.h>
#include <omp.h>


using namespace std;
using namespace thrust;

struct calculate_score
{
    char letter;
    thrust::device_ptr<char> data_subsequence;
    thrust::device_ptr<int> last_line_calculation;

    calculate_score(thrust::device_ptr<char> _data_subsequence, char _letter, thrust::device_ptr<int>_last_line_calculation) : data_subsequence(_data_subsequence), letter(_letter), last_line_calculation(_last_line_calculation) {};

    __host__ __device__
    int operator()(const int& i) {

        int diagonal = 0;
        int insertion = last_line_calculation[i] - 1;

        if (data_subsequence[i-1] == letter) {
            diagonal = last_line_calculation[i-1] + 2;
        } else {
            diagonal = last_line_calculation[i-1] - 1;
        }

        if (diagonal > insertion) {
            if (diagonal > 0) {
                return diagonal;
            } else {
                return 0;
            }
        } else {
            if (insertion > 0) {
                return insertion;
            } else {
                return 0;
            }
        }
    }
};

int main() {

    double n = 0;
    double m = 0;

    cin >> n >> m;

    host_vector<char> dnas_first_seq_cpu;
    host_vector<char> dnas_second_seq_cpu;

    dnas_first_seq_cpu.reserve(n);
    dnas_second_seq_cpu.reserve(m);

    char dna_next;

    // Reads the first sequence and completes the dna_first_seq vector with the sequence.
    for(int i = 0; i < n; i++) {
        cin >> dna_next;
        dnas_first_seq_cpu.push_back(dna_next);
    }

    // Reads the second sequence and completes the dna_second_seq vector with the sequence.
    for(int i = 0; i < m; i++) {
        cin >> dna_next;
        dnas_second_seq_cpu.push_back(dna_next);
    }
    
    int k = n;
    int l = m;

    host_vector<char> dnas_first_seq_cpu_flexible = dnas_first_seq_cpu;
    host_vector<char> dnas_second_seq_cpu_flexible = dnas_second_seq_cpu;

    host_vector<host_vector<char> > subsequences_a;
    host_vector<host_vector<char> > subsequences_b;

    while (dnas_first_seq_cpu_flexible.size() > 0) {
        while(k > 0) {
            host_vector<char> next_subsequence;
            for (int j = 0; j < k; j++) {
                next_subsequence.push_back(dnas_first_seq_cpu_flexible[j]);
            }
            k = k - 1;

            subsequences_a.push_back(next_subsequence);
            next_subsequence.clear();
        }
        dnas_first_seq_cpu_flexible.erase(dnas_first_seq_cpu_flexible.begin());
        k = dnas_first_seq_cpu_flexible.size();
    }

    while (dnas_second_seq_cpu_flexible.size() > 0) {
        while(l > 0) {
            host_vector<char> next_subsequence;
            for (int j = 0; j < l; j++) {
                next_subsequence.push_back(dnas_second_seq_cpu_flexible[j]);
            }
            l = l - 1;

            subsequences_b.push_back(next_subsequence);
            next_subsequence.clear();
        }
        dnas_second_seq_cpu_flexible.erase(dnas_second_seq_cpu_flexible.begin());
        l = dnas_second_seq_cpu_flexible.size();
    }

    int max_score = 0;


    for (int i = 0; i < subsequences_a.size(); i++) {
        char *subsequence_from_sequence_a = subsequences_a[i].data();
        device_vector<char> subsequence_a_gpu = subsequences_a[i];

        for (int j = 0; j < subsequences_b.size(); j++) {
            char *subsequence_from_sequence_b = subsequences_a[j].data();
            device_vector<char> subsequence_b_gpu = subsequences_b[j];
            device_vector<int> calculation_temp(subsequences_a[i].size() + 1);
            device_vector<int> calculation_score(subsequences_a[i].size() + 1);

            thrust::fill(calculation_temp.begin(), calculation_temp.end(), 0);

            #pragma omp parallel
            {
                // Creating tasks for each sequence to create subsequences.
                #pragma omp master
                {
                    #pragma omp task
                    {
                        for (int t = 0; t < subsequences_b[j].size(); t++) {

                            char letter_from_subsequence_b = subsequences_b[j][t];

                            thrust::device_vector<int> counting(subsequence_a_gpu.size() + 1);

                            for (int q = 0; q < subsequences_a[i].size() + 1; q++) {
                                counting[q] = q;
                            }

                            thrust::transform(counting.begin(), counting.end(), calculation_score.begin(), calculate_score(subsequence_a_gpu.data(), letter_from_subsequence_b, calculation_temp.data()));
                            thrust::inclusive_scan(calculation_score.begin()+1, calculation_score.end(), calculation_temp.begin() + 1, thrust::maximum<int>());

                            int max = *max_element(calculation_temp.begin(), calculation_temp.end());

                            cout << max;
                            cout << endl;
                            cout << max_score;

                            if (max > max_score) {
                                max_score = max;
                                cout << max_score;

                            }
                        }                    
                    }
                }
            }
        }
    }

    cout << max_score;
}